#include "hip/hip_runtime.h"
#include"Demosaicing.cuh"

__global__ void processDemosacing(unsigned char* device_bayer_data, unsigned char* device_bgr_data, int width, int height)
{
    unsigned long long x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        unsigned long long index = y * width + x;
        unsigned char r, g, b;

        // 根据拜尔模式（RGGB）进行插值
        if (x % 2 == 0 && y % 2 == 0) // R
        {
            r = device_bayer_data[index];
            g = (device_bayer_data[index - 1] + device_bayer_data[index + 1] + device_bayer_data[index - width] + device_bayer_data[index + width]) / 4;
            b = (device_bayer_data[index - width - 1] + device_bayer_data[index - width + 1] + device_bayer_data[index + width - 1] + device_bayer_data[index + width + 1]) / 4;
        }
        else if (x % 2 == 1 && y % 2 == 0) // Gr
        {
            r = (device_bayer_data[index - 1] + device_bayer_data[index + 1]) / 2;
            g = device_bayer_data[index];
            b = (device_bayer_data[index - width] + device_bayer_data[index + width] + device_bayer_data[index - width - 1] + device_bayer_data[index - width + 1] + device_bayer_data[index + width - 1] + device_bayer_data[index + width + 1]) / 6;
        }
        else if (x % 2 == 0 && y % 2 == 1) // Gb
        {
            r = (device_bayer_data[index - width] + device_bayer_data[index + width]) / 2;
            g = device_bayer_data[index];
            b = (device_bayer_data[index - 1] + device_bayer_data[index + 1] + device_bayer_data[index - width - 1] + device_bayer_data[index - width + 1] + device_bayer_data[index + width - 1] + device_bayer_data[index + width + 1]) / 6;
        }
        else // B
        {
            r = (device_bayer_data[index - width - 1] + device_bayer_data[index - width + 1] + device_bayer_data[index + width - 1] + device_bayer_data[index + width + 1]) / 4;
            g = (device_bayer_data[index - 1] + device_bayer_data[index + 1] + device_bayer_data[index - width] + device_bayer_data[index + width]) / 4;
            b = device_bayer_data[index];
        }

        // 存储结果到BGR图像数组
        int bgr_index = index * 3;
        device_bgr_data[bgr_index] = b;
        device_bgr_data[bgr_index + 1] = g;
        device_bgr_data[bgr_index + 2] = r;
    }
}
