#include "hip/hip_runtime.h"
#include"Demosaicing.cuh"

__global__  void processDemosacing(unsigned char* device_bayer_data, unsigned char* device_bgr_data, int width, int height);
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    int bayer_idx = y * width + x;
    int bgr_idx = bayer_idx * 3;

    if (y % 2 == 0) {
        if (x % 2 == 0)
        {
            // B
            device_bgr_data[bgr_idx + 0] = device_bayer_data[bayer_idx];
            device_bgr_data[bgr_idx + 1] = (x + 1 < width) ? device_bayer_data[bayer_idx + 1] : device_bayer_data[bayer_idx - 1];
            device_bgr_data[bgr_idx + 2] = (y + 1 < height) ? device_bayer_data[bayer_idx + width] : device_bayer_data[bayer_idx - width];
        } 
        else 
        {
            // G
            device_bgr_data[bgr_idx + 0] = (x - 1 >= 0) ? device_bayer_data[bayer_idx - 1] : device_bayer_data[bayer_idx + 1];
            device_bgr_data[bgr_idx + 1] = device_bayer_data[bayer_idx];
            device_bgr_data[bgr_idx + 2] = (y + 1 < height) ? device_bayer_data[bayer_idx + width] : device_bayer_data[bayer_idx - width];
        }
    } 
}
