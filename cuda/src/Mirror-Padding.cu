#include<iostream>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

void mirrorPadBayerImage(const std::vector<unsigned char>& bayer_data, std::vector<unsigned char>& padded_data, int width, int height, int pad_width) {
    int padded_width = width + 2 * pad_width;
    int padded_height = height + 2 * pad_width;
    padded_data.resize(padded_width * padded_height, 0);
    for (int y = 0; y < height; ++y) 
    {
        for (int x = 0; x < width; ++x) 
        {
            int src_idx = y * width + x;
            int dst_idx = (y + pad_width) * padded_width + (x + pad_width);
            padded_data[dst_idx] = bayer_data[src_idx];
        }
    }
    for (int y = 0; y < pad_width; ++y)
     {
        for (int x = 0; x < padded_width; ++x)
         {
            padded_data[y * padded_width + x] = padded_data[(pad_width * 2 - y - 1) * padded_width + x];
            padded_data[(padded_height - y - 1) * padded_width + x] = padded_data[(padded_height - pad_width * 2 + y) * padded_width + x];
        }
    }
    for (int y = pad_width; y < padded_height - pad_width; ++y) 
    {
        for (int x = 0; x < pad_width; ++x) 
        {
            padded_data[y * padded_width + x] = padded_data[y * padded_width + (pad_width * 2 - x - 1)];
            padded_data[y * padded_width + (padded_width - x - 1)] = padded_data[y * padded_width + (padded_width - pad_width * 2 + x)];
        }
    }
}