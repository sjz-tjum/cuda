#include"set_GPU.cuh"


void set_GPU()
{
    int iDeviceCount = 0;
    hipError_t error = hipGetDeviceCount( &iDeviceCount);
    if(error != hipSuccess)
    {
        printf("There's no GPU found!\n");
        exit(-1);
    }
    else
    {
        printf("The number of GPU is %d .\n",iDeviceCount);
    }

   //设置GPU
   int iDev = 0;
   error = hipSetDevice(iDev);
   if(error != hipSuccess)
   {
    printf("fail to set GPU 0 for computing.\n");
    exit(-1);
   }
   else
   {
    printf("Successfully set GPU 0 for computing.\n");
   }
}