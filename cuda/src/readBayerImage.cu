#include "readBayer.cuh"
#include<iostream>


std::vector<unsigned char> readBayer_img::readBayerImage(const std::string& filename, int width, int height)
{
    std::ifstream file(filename, std::ios::binary);   //文件以二进制模式打开
    if (!file) {
        std::cerr << "Error: Could not open or find the image" << std::endl;
        exit(1);
    }
    std::vector<unsigned char> bayer_data(width * height);
    file.read(reinterpret_cast<char*>(bayer_data.data()), width * height);
    return bayer_data;
}