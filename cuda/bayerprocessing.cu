#include <iostream>
#include <hip/hip_runtime.h> 
#include <opencv2/opencv.hpp>
#include "common.cuh"
#include  "readBayer.cuh"
#include  "Mirror-Padding.cu"


int main(void)
{
  set_GPU();  
  std::cout<<"请输入图片的长和宽"<<std::endl;
  int width,height;
  std::cin>>width>>height;
  readBayer_img Bayer{"example",width,height};
  std::vector<unsigned char> Bayer_data_ = Bayer.readBayerImage("example",width,height);
  std::vector<unsigned char> BGR_data_;
  std::vector<unsigned char>& padded_data;
  int pad_width = 10;
  int padded_width = width + pad_width;
  int padded_height = height + pad_width;
  mirrorPadBayerImage(Bayer_data_ ,padded_data,width,height,pad_width);
  //分配主机内存，并初始化
  int Bayer_ElemenCount = ( width + pad_width ) * ( height + pad_width ) ;
  int BGR_ElementCount = 3 * width * height ;

  size_t Bayer_Bytecount = Bayer_ElemenCount*sizeof(unsigned char);
  size_t Bgr_Bytecount = BGR_ElementCount * sizeof(unsigned char);

  unsigned char *device_padded_bayer_data; unsigned char*device_bgr_data;
  device_bayer_data = (float *)hipMalloc(Bayer_Bytecount);
  device_bgr_data = (float *)hipMalloc(Bgr_Bytecount);
  //复制主机数据到GPU
  hipMemcpy(device_padded_bayer_data, padded_bayer_data.data(), padded_width * padded_height * sizeof(unsigned char), hipMemcpyHostToDevice);
  //设置网格数和线程数
  dim3 threadsPerBlock(16, 16);
  dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
  
  Demosaicing_process<<<blocksPerGrid, threadsPerBlock>>>(device_padded_bayer_data, device_bgr_data, padded_width, padded_height, width, height, pad_width);
  hipDeviceSynchronize();

  std::vector<unsigned char> bgr_data(width * height * 3);
  hipMemcpy(bgr_data.data(), device_bgr_data, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(device_padded_bayer_data);
  hipFree(device_bgr_data);

  
  cv::Mat bgr_image(height, width, CV_8UC3, bgr_data.data());
  cv::imwrite("output.jpg", bgr_image);

  return 0;  
}

