#include <iostream>
#include <hip/hip_runtime.h>  // 包含CUDA运行时头文件
#include <opencv2/opencv.hpp>
#include "common.cuh"


int main(void)
{
  set_GPU();  
  std::cout<<"请输入图片的长和宽"<<std::endl;
  std::cin>>width>>height;
  readBayer_img Bayer{"example",width,height};
  std::vector<unsigned char> Bayer_data_ = Bayer.readBayerImage("example",width,height);
  std::vector<unsigned char> BGR_data_;
  //分配主机内存，并初始化
  int Bayer_ElemenCount = width*height;
  int BGR_ElementCount = 3*width*height;

  size_t Bayer_Bytecount = Bayer_ElemenCount*sizeof(unsigned char);
  size_t Bgr_Bytecount = BGR_ElementCount * sizeof(unsigned char);


  unsigned char *device_bayer_data; unsigned char*device_bgr_data;
  device_bayer_data = (float *)malloc(Bayer_Bytecount);
  device_bgr_data = (float *)malloc(Bgr_Bytecount);
  //复制主机数据到GPU
  hipMemcpy(device_bayer_data,Bayer_data_.data(), Bayer_Bytecount, hipMemcpyHostToDevice);
  //设置网格数和线程数
  dim3 threadsPerBlock(16, 16);
  dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);



}