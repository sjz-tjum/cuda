#include<iostream>
#include<stdlib.h>
#include <hip/hip_runtime.h>
//#include <opencv2/opencv.hpp>

__global__ void Demosaicing_process(unsigned char* padded_bayer_data, unsigned char* bgr_data, int padded_width, int padded_height, int width, int height,int pad_width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x + pad_width;  
    int y = blockIdx.y * blockDim.y + threadIdx.y + pad_width;  

    if (x >= width + pad_width || y >= height + pad_width) return;
    int bayer_idx = y * padded_width + x;
    int bgr_idx = (y - pad_width) * width * 3 + (x - pad_width) * 3;
    unsigned char r, g, b;
    if (x % 2 == 0 && y % 2 == 0) { 
        r = padded_bayer_data[bayer_idx];
        g = (padded_bayer_data[bayer_idx - 1] + padded_bayer_data[bayer_idx + 1] + padded_bayer_data[bayer_idx - padded_width] + padded_bayer_data[bayer_idx + padded_width]) / 4;
        b = (padded_bayer_data[bayer_idx - padded_width - 1] + padded_bayer_data[bayer_idx - padded_width + 1] + padded_bayer_data[bayer_idx + padded_width - 1] + padded_bayer_data[bayer_idx + padded_width + 1]) / 4;
    } else if (x % 2 == 1 && y % 2 == 0) { 
        r = (padded_bayer_data[bayer_idx - 1] + padded_bayer_data[bayer_idx + 1]) / 2;
        g = padded_bayer_data[bayer_idx];
        b = (padded_bayer_data[bayer_idx - padded_width] + padded_bayer_data[bayer_idx + padded_width] + padded_bayer_data[bayer_idx - padded_width - 1] + padded_bayer_data[bayer_idx - padded_width + 1] + padded_bayer_data[bayer_idx + padded_width - 1] + padded_bayer_data[bayer_idx + padded_width + 1]) / 6;
    } else if (x % 2 == 0 && y % 2 == 1) { 
        r = (padded_bayer_data[bayer_idx - padded_width] + padded_bayer_data[bayer_idx + padded_width]) / 2;
        g = padded_bayer_data[bayer_idx];
        b = (padded_bayer_data[bayer_idx - 1] + padded_bayer_data[bayer_idx + 1] + padded_bayer_data[bayer_idx - padded_width - 1] + padded_bayer_data[bayer_idx - padded_width + 1] + padded_bayer_data[bayer_idx + padded_width - 1] + padded_bayer_data[bayer_idx + padded_width + 1]) / 6;
    } else { // B
        r = (padded_bayer_data[bayer_idx - padded_width - 1] + padded_bayer_data[bayer_idx - padded_width + 1] + padded_bayer_data[bayer_idx + padded_width - 1] + padded_bayer_data[bayer_idx + padded_width + 1]) / 4;
        g = (padded_bayer_data[bayer_idx - 1] + padded_bayer_data[bayer_idx + 1] + padded_bayer_data[bayer_idx - padded_width] + padded_bayer_data[bayer_idx + padded_width]) / 4;
        b = padded_bayer_data[bayer_idx];
    }
    bgr_data[bgr_idx] = b;
    bgr_data[bgr_idx + 1] = g;
    bgr_data[bgr_idx + 2] = r;
}