#include <iostream>
#include <string>
#include <vector>
#include <fstream>
//#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h> 
#include "C:\Users\ASUS\Desktop\cuda\src\Mirror-Padding.cu"
#include "C:\Users\ASUS\Desktop\cuda\src\processingD.cu"
#include "C:\Users\ASUS\Desktop\cuda\src\set_GPU.cu"


std::vector<unsigned char> readBayerImage(const std::string& filename, int width, int height) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error: Could not open or find the image" << std::endl;
        exit(1);
    }

    std::vector<unsigned char> bayer_data(width * height);
    file.read(reinterpret_cast<char*>(bayer_data.data()), width * height);
    if (!file) {
        std::cerr << "Error: Failed to read the image" << std::endl;
        exit(1);
    }

    return bayer_data;
}

int main(void) {
    set_GPU();  
    std::string filename = "C:/Users/ASUS/Desktop/cuda/examples/example.RAW";
    int width = 7040; 
    int height = 4688;

    // 读取Bayer图像数据
    std::vector<unsigned char> Bayer_data_ = readBayerImage(filename, width, height);

    // 初始化padded_data
    std::vector<unsigned char> padded_data;

    int pad_width = 10;
    int padded_width = width + 2 * pad_width;
    int padded_height = height + 2 * pad_width;

    // 进行镜像填充
    mirrorPadBayerImage(Bayer_data_, padded_data, width, height, pad_width);

    // 分配GPU内存
    unsigned char* device_padded_bayer_data;
    unsigned char* device_bgr_data;

    int Bayer_ElemenCount = padded_width * padded_height;
    int BGR_ElementCount = 3 * width * height;

    size_t Bayer_Bytecount = Bayer_ElemenCount * sizeof(unsigned char);
    size_t Bgr_Bytecount = BGR_ElementCount * sizeof(unsigned char);

    hipMalloc((void**)&device_padded_bayer_data, Bayer_Bytecount);
    hipMalloc((void**)&device_bgr_data, Bgr_Bytecount);

    // 复制主机数据到GPU
    hipMemcpy(device_padded_bayer_data, padded_data.data(), Bayer_Bytecount, hipMemcpyHostToDevice);

    // 设置网格数和线程数
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 记录开始时间
    hipEventRecord(start, 0);
    
    // 调用核函数进行去马赛克处理
    Demosaicing_process<<<blocksPerGrid, threadsPerBlock>>>(device_padded_bayer_data, device_bgr_data, padded_width, padded_height, width, height, pad_width);
    hipDeviceSynchronize();
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // 计算并输出执行时间
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Kernel execution time: " << elapsedTime << " ms" << std::endl;

    // 将结果从设备复制回主机
    std::vector<unsigned char> bgr_data(width * height * 3);
    hipMemcpy(bgr_data.data(), device_bgr_data, Bgr_Bytecount, hipMemcpyDeviceToHost);

    // 释放GPU内存
    hipFree(device_padded_bayer_data);
    hipFree(device_bgr_data);

    // 保存或处理BGR图像数据
   //  cv::Mat bgr_image(height, width, CV_8UC3, bgr_data.data());
    // cv::imwrite("output.jpg", bgr_image);
   // cv::imshow("BGR Image", bgr_image);
    // cv::waitKey(0);  // 等待按键
    std::cout << "Original Bayer Image Data (part):" << std::endl;
    int rows_to_print = 5;  // 打印的行数
    int cols_to_print = 5;  // 打印的列数
    for (int y = 0; y < rows_to_print; ++y) {
        for (int x = 0; x < cols_to_print; ++x) {
            int idx = y * width + x;
            std::cout << static_cast<int>(Bayer_data_[idx]) << " ";
        }
        std::cout << std::endl;
    }

    // 打印BGR图像数据矩阵的一部分
    std::cout << "Processed BGR Image Data (part):" << std::endl;
    for (int y = 0; y < rows_to_print; ++y) {
        for (int x = 0; x < cols_to_print; ++x) {
            int idx = (y * width + x) * 3;
            std::cout << "B: " << static_cast<int>(bgr_data[idx]) 
                      << " G: " << static_cast<int>(bgr_data[idx + 1]) 
                      << " R: " << static_cast<int>(bgr_data[idx + 2]) 
                      << "  ";
        }
        std::cout << std::endl;
    }

    return 0;  
}
